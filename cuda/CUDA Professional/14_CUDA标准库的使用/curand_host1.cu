
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>

#include <cstdio>
#include <cstdlib>

void output_results(int N, double *g_x);

int main(int argc, char *argv[]) {
    hiprandGenerator_t generator;
    hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(generator, 1234);
    int N = 100000;
    double *g_x;
    hipMalloc((void **)&g_x, sizeof(double) * N);
    hiprandGenerateUniformDouble(generator, g_x, N);
    double *x = (double *)calloc(N, sizeof(double));
    hipMemcpy(x, g_x, sizeof(double) * N, hipMemcpyDeviceToHost);
    hipFree(g_x);
    output_results(N, x);

    free(x);

    return 0;
}

void output_results(int N, double *x) {
    FILE *fid = fopen("x1.txt", "w");
    for (int n = 0; n < N; n++) {
        fprintf(fid, "%g\n", x[n]);
    }
    fclose(fid);
}
