#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "hip/hip_cooperative_groups.h"
//#define THREAD_PER_BLOCK 256
// 注意：
// 1.本节的文字解析放在了CUDA_lesson.pdf，如有不懂，可以先看看文字解析
// 2.这种warp和shared在老的gpu上面会很有成效，但是在turing后的GPU，nvcc编译器优化了很多，所以导致效果不明显
// 3.我记得在某个CUDA版本之前，atomic是可以保证block或thread严格按照ID串行，但是某个CUDA版本之后，就不行了，至少在现有流行版本不行了，所以会发现CUDA copy if执行后，虽然全都是>0的值，但是顺序和输入不一样
// cpu实现
int filter(int *dst, int *src, int n) {
  int nres = 0;
  for (int i = 0; i < n; i++)
    if (src[i] > 0)
      dst[nres++] = src[i];
  // return the number of elements copied
  return nres;
}
// 数据量为256000000时，latency=14.37ms
// naive kernel
//__global__ void filter_k(int *dst, int *nres, int *src, int n) {
//  int i = threadIdx.x + blockIdx.x * blockDim.x;
//  // 输入数据大于0的，计数器+1，并把该数写到输出显存以计数器值为索引的地址
//  if(i < n && src[i] > 0)
//    dst[atomicAdd(nres, 1)] = src[i];
//}

// 数据量为256000000时，latency=13.86ms
// block level, use block level atomics based on shared memory
// __global__ 
// void filter_shared_k(int *dst, int *nres, const int* src, int n) {
//   // 计数器声明为shared memory，去计数各个block范围内大于0的数量
//   __shared__ int l_n;
//   int gtid = blockIdx.x * blockDim.x + threadIdx.x;
//   int total_thread_num = blockDim.x * gridDim.x;

//   for (int i = gtid; i < n; i += total_thread_num) {
//     // use first thread to zero the counter
//     // 初始化只需1个线程来操作
//     if (threadIdx.x == 0)
//       l_n = 0;
//     __syncthreads();

//     int d, pos;
//     // l_n表示每个block范围内大于0的数量，block内的线程都可访问
//     // pos是每个线程私有的寄存器，且作为atomicAdd的返回值，表示当前线程对l_n原子加1之前的l_n，比如1 2 4号线程都大于0，那么对于4号线程来说l_n = 3, pos = 2
//     if(i < n && src[i] > 0) {
//         pos = atomicAdd(&l_n, 1);
//     }
//     __syncthreads();

//     // 每个block选出tid=0作为leader
//     // leader把每个block的l_n累加到全局计数器(nres),即所有block的局部计数器做一个reduce sum
//     // 注意: 下下行原子加返回的l_n为全局计数器nres原子加l_n之前的nres，比如对于block1，已知原子加前，nres = 2, l_n = 3，原子加后, nres = 2+3, 返回的l_n = 2
//     if(threadIdx.x == 0)
//       l_n = atomicAdd(nres, l_n);
//     __syncthreads();

//     //write & store
//     if(i < n && d > 0) {
//     // 1. pos: src[thread]>0的thread在当前block的index
//     // 2. l_n: 在当前block的前面几个block的所有src>0的个数
//     // 3. pos + l_n：当前thread的全局offset
//       pos += l_n; 
//       dst[pos] = d;
//     }
//     __syncthreads();
//   }
// }

//数据量为256000000时，latency=13.79ms
//warp level, use warp-aggregated atomics
__device__ int atomicAggInc(int *ctr) {
  unsigned int active = __activemask();
  int leader = __ffs(active) - 1; // 视频所示代码这里有误，leader应该表示warp里面第一个src[threadIdx.x]>0的threadIdx.x
  int change = __popc(active);//warp mask中为1的数量
  int lane_mask_lt;
  asm("mov.u32 %0, %%lanemask_lt;" : "=r"(lane_mask_lt));
  unsigned int rank = __popc(active & lane_mask_lt); // 比当前线程id小且值为1的mask之和
  int warp_res;
  if(rank == 0)//leader thread of every warp
    warp_res = atomicAdd(ctr, change);//compute global offset of warp
  warp_res = __shfl_sync(active, warp_res, leader);//broadcast warp_res of leader thread to every active thread
  return warp_res + rank; // global offset + local offset = final offset，即L91表示的atomicAggInc(nres), 为src[i]的最终的写入到dst的位置
}

__global__ void filter_warp_k(int *dst, int *nres, const int *src, int n) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if(i >= n)
    return;
  if(src[i] > 0) // 过滤出src[i] > 0 的线程，比如warp0里面只有0号和1号线程的src[i]>0，那么只有0号和1号线程运行L91，对应的L72的__activemask()为110000...00
    // 以上L71函数计算当前thread负责数据的全局offset
    dst[atomicAggInc(nres)] = src[i];
}

bool CheckResult(int *out, int groudtruth, int n){
    if (*out != groudtruth) {
        return false;
    }
    return true;
}

int main(){
    float milliseconds = 0;
    int N = 2560000;

    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    const int blockSize = 256;
    int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);

    int *src_h = (int *)malloc(N * sizeof(int));
    int *dst_h = (int *)malloc(N * sizeof(int));
    int *nres_h = (int *)malloc(1 * sizeof(int));
    int *dst, *nres;
    int *src;
    hipMalloc((void **)&src, N * sizeof(int));
    hipMalloc((void **)&dst, N * sizeof(int));
    hipMalloc((void **)&nres, 1 * sizeof(int));

    for(int i = 0; i < N; i++){
        src_h[i] = 1;
    }

    int groudtruth = 0;
    for(int j = 0; j < N; j++){
        if (src_h[j] > 0) {
            groudtruth += 1;
        }
    }


    hipMemcpy(src, src_h, N * sizeof(int), hipMemcpyHostToDevice);

    dim3 Grid(GridSize);
    dim3 Block(blockSize);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    filter_warp_k<<<Grid, Block>>>(dst, nres, src, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(nres_h, nres, 1 * sizeof(int), hipMemcpyDeviceToHost);
    bool is_right = CheckResult(nres_h, groudtruth, N);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        printf("%lf ",*nres_h);
        printf("\n");
    }
    printf("filter_k latency = %f ms\n", milliseconds);    

    hipFree(src);
    hipFree(dst);
    hipFree(nres);
    free(src_h);
    free(dst_h);
    free(nres_h);
}
