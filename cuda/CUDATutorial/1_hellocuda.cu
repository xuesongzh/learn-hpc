#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void hello_cuda(){
    // 泛指当前线程在所有block范围内的全局id
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("block id = [ %d ], thread id = [ %d ] hello cuda\n", blockIdx.x, idx);
}

int main() {
    hello_cuda<<< 1, 1 >>>();
    hipDeviceSynchronize();
    return 0;
}