#include <stdio.h>
#include <hip/hip_runtime.h>


typedef float FLOAT;

/* CUDA kernel function */
__global__ void vec_add(FLOAT *x, FLOAT *y, FLOAT *z, int N)
{
    /* 2D grid */
    int idx = (blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x);
    /* 1D grid */
    // int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) z[idx] = y[idx] + x[idx];
}

void vec_add_cpu(FLOAT *x, FLOAT *y, FLOAT *z, int N)
{
    for (int i = 0; i < N; i++) z[i] = y[i] + x[i];
}

int main()
{
    int N = 10000;
    int nbytes = N * sizeof(FLOAT);

    /* 1D block */
    int bs = 256;

    /* 2D grid */
    int s = ceil(sqrt((N + bs - 1.) / bs));
    dim3 grid(s, s);
    /* 1D grid */
    // int s = ceil((N + bs - 1.) / bs);
    // dim3 grid(s);

    FLOAT *dx, *hx;
    FLOAT *dy, *hy;
    FLOAT *dz, *hz;

    /* allocate GPU mem */
    hipMalloc((void **)&dx, nbytes);
    hipMalloc((void **)&dy, nbytes);
    hipMalloc((void **)&dz, nbytes);
    
    /* init time */
    float milliseconds = 0;

    /* alllocate CPU mem */
    hx = (FLOAT *) malloc(nbytes);
    hy = (FLOAT *) malloc(nbytes);
    hz = (FLOAT *) malloc(nbytes);

    /* init */
    for (int i = 0; i < N; i++) {
        hx[i] = 1;
        hy[i] = 1;
    }

    /* copy data to GPU */
    hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(dy, hy, nbytes, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    /* launch GPU kernel */
    vec_add<<<grid, bs>>>(dx, dy, dz, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);  
    
	/* copy GPU result to CPU */
    hipMemcpy(hz, dz, nbytes, hipMemcpyDeviceToHost);

    /* CPU compute */
    FLOAT* hz_cpu_res = (FLOAT *) malloc(nbytes);
    vec_add_cpu(hx, hy, hz_cpu_res, N);

    /* check GPU result with CPU*/
    for (int i = 0; i < N; ++i) {
        if (fabs(hz_cpu_res[i] - hz[i]) > 1e-6) {
            printf("Result verification failed at element index %d!\n", i);
        }
    }
    printf("Result right\n");
    hipFree(dx);
    hipFree(dy);
    hipFree(dz);

    free(hx);
    free(hy);
    free(hz);
    free(hz_cpu_res);

    return 0;
}