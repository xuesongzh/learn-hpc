#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


// v0: naive版本
// latency: 3.835ms
// blockSize作为模板参数的效果主要用于静态shared memory的申请需要传入编译期常量指定大小（L10)
template<int blockSize>
__global__ void reduce_v0(float *d_in,float *d_out){
    __shared__ float smem[blockSize];
    // 泛指当前线程在其block内的id
    int tid = threadIdx.x;
    // 泛指当前线程在所有block范围内的全局id
    int gtid = blockIdx.x * blockSize + threadIdx.x;
    // load: 每个线程加载一个元素到shared mem对应位置
    smem[tid] = d_in[gtid];
    // 涉及到对shared memory的读写最好都加上__syncthreads
    __syncthreads();

    // 每个线程在shared memory上跨index加另一个元素，直到跨度>线程数量
    // 此时一个block对d_in这块数据的reduce sum结果保存在id为0的线程上面
    for(int index = 1; index < blockDim.x; index *= 2) {
        // 注意！v0并没有warp divergence，因为没有else分支，视频目前这里讲错
        // 现在的v0和v1性能大体相似
        // v0慢的原因在于下一行使用了除余%，除余%是个非常耗时的指令，我会在下个版本对这里进一步修正
        // 可尝试把下一行替换为`if ((tid & (2 * index - 1)) == 0) {`, 性能大概可以提升30%～50%
        if (tid % (2 * index) == 0) {
            smem[tid] += smem[tid + index];
        }
        __syncthreads();
    }

    // store: 哪里来回哪里去，把reduce结果写回显存
    if (tid == 0) {
        d_out[blockIdx.x] = smem[0];
    }
}
bool CheckResult(float *out, float groudtruth, int n){
    float res = 0;
    for (int i = 0; i < n; i++){
        res += out[i];
    }
    if (res != groudtruth) {
        return false;
    }
    return true;
}

int main(){
    float milliseconds = 0;
    const int N = 25600000;
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    const int blockSize = 256;
    int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
    //int GridSize = 100000;
    float *a = (float *)malloc(N * sizeof(float));
    float *d_a;
    hipMalloc((void **)&d_a, N * sizeof(float));

    float *out = (float*)malloc((GridSize) * sizeof(float));
    float *d_out;
    hipMalloc((void **)&d_out, (GridSize) * sizeof(float));

    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
    }

    float groudtruth = N * 1.0f;

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(GridSize);
    dim3 Block(blockSize);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_v0<blockSize><<<Grid,Block>>>(d_a, d_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(out, d_out, GridSize * sizeof(float), hipMemcpyDeviceToHost);
    printf("allcated %d blocks, data counts are %d", GridSize, N);
    bool is_right = CheckResult(out, groudtruth, GridSize);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        //for(int i = 0; i < GridSize;i++){
            //printf("res per block : %lf ",out[i]);
        //}
        //printf("\n");
        printf("groudtruth is: %f \n", groudtruth);
    }
    printf("reduce_v0 latency = %f ms\n", milliseconds);

    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);
}
