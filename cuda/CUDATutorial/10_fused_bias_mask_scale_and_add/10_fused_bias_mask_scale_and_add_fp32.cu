#include <hip/hip_runtime.h>
#include <bits/stdc++.h>


// 实现fp32的fused biasadd mask scale and add的融合算子
// biasadd + mask + scale + elemwise_add四个算子的融合
// （x + bias） * mask * scale + addend;

template<typename T>
struct MaskScaleAndElemwiseAddFunctor
{
    // 有参构造函数
    MaskScaleAndElemwiseAddFunctor(const uint8_t * mask, const T * add_val, float scale)
    :_mask(mask), _add_val(add_val), _scale(scale)
    {}

    // 重载运算符（）
    __device__ T operator()(T x, int i) const
    {
        return x * static_cast<T>(static_cast<bool>(_mask[i]) * _scale) + _add_val[i];
    }

    const uint8_t * _mask;
    const T * _add_val;
    float _scale;
};
// 朴素写法：和视频上的一致
template<int biasSize, typename FUNCTOR, typename T>
__global__ void FusedBaisAdd(FUNCTOR functor, T * dx, T * dy, T * d_bias, const int n, const int bias_size)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    // 对比于59行的读向量，此处读标量，总数为元素个数n
    for (int i = gid; i < n; i += gridDim.x * blockDim.x)
    {
        // 先加上bias
        T tmp = dx[i] + d_bias[i % bias_size];
        // 再做mask+scale+elementwiseadd
        dy[i] = functor(tmp, i);
    }
}

// 使用向量化进行存取
template<int biasSize, typename FUNCTOR, typename T>
__global__ void FusedBaisAddVecSmem(FUNCTOR functor, T * dx, T * dy, T * d_bias, const int n, const int bias_size)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    int tid = threadIdx.x;

    __shared__ T smem[biasSize];

    // 将d_bias放在shared memory上
    // 原因: d_bias在62-65行通过除余bias_size被多次读取复用，所以放在shared memory上
    if (tid < bias_size)
        smem[tid] = d_bias[tid];
    __syncthreads();
    // float4向量化读取，此时1个线程读取4个数，因此需要读取的向量总数为n/4
    for (int i = gid; i < n / 4; i += gridDim.x * blockDim.x)
    {
        float4 a = reinterpret_cast<float4 *>(dx)[i];
        float4 b;

        b.x = functor(a.x + smem[(i * 4) % bias_size], i * 4);
        b.y = functor(a.y + smem[(i * 4 + 1) % bias_size], i * 4 + 1);
        b.z = functor(a.z + smem[(i * 4 + 2) % bias_size], i * 4 + 2);
        b.w = functor(a.w + smem[(i * 4 + 3) % bias_size], i * 4 + 3);

        reinterpret_cast<float4*>(dy)[i] = b;
    }
}

bool CheckRight(float * y, float * groudTruth, const int n)
{
    for (int i = 0; i < n; ++i)
    {
        if (y[i] != groudTruth[i])
        {
            printf("y[%d] : %f \n", i, y[i]);
            printf("groundTruth[%d] : %f\n", i, groudTruth[i]);
            return false;
        }
    }
    return true;
}

int main()
{
    constexpr int n = 100000;
    constexpr int bias_size = 10;
    
    float scale = 0.5;
    uint8_t * mask_tensor = new uint8_t[n];
    float * add_val = new float[n];
    // 初始化
    for (int i = 0; i < n; ++i)
    {
        mask_tensor[i] = (uint8_t)(i);
        add_val[i] = (float)(i);
    }

    float * x = (float *)malloc(sizeof(float) * n);
    float * y = (float *)malloc(sizeof(float) * n);
    float * bias = (float *)malloc(sizeof(float) * bias_size);
    for (int i = 0; i < n; ++i)
    {
        x[i] = (float)(i);
        y[i] = 0.0f;
    }
    for (int i = 0; i < bias_size; ++i)
        bias[i] = i;

    float * groudTruth = (float *)malloc(sizeof(float) * n);
    for (int i = 0; i < n; ++i)
    {
        groudTruth[i] = (x[i] + bias[i % bias_size]) * static_cast<float>(static_cast<bool>(mask_tensor[i]) * scale) + add_val[i];
    }

    float * dx, * dy, * d_bias;
    hipMalloc((void **)&dx, sizeof(float) * n);
    hipMalloc((void **)&dy, sizeof(float) * n);
    hipMalloc((void **)&d_bias, sizeof(float) * bias_size);
    hipMemcpy(dx, x, sizeof(float) * n, hipMemcpyHostToDevice);
    hipMemcpy(dy, y, sizeof(float) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_bias, bias, sizeof(float) * bias_size, hipMemcpyHostToDevice);
    uint8_t * d_mask_tensor;
    float * d_add_val;
    hipMalloc((void **)&d_mask_tensor, sizeof(uint8_t) * n);
    hipMalloc((void **)&d_add_val, sizeof(float) * n);
    hipMemcpy(d_mask_tensor, mask_tensor, sizeof(uint8_t) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_add_val, add_val, sizeof(float) * n, hipMemcpyHostToDevice);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    int blockSize = 512;
    int gridSize = std::min((n + blockSize - 1) / blockSize, deviceProp.maxGridSize[0]);

    MaskScaleAndElemwiseAddFunctor<float> functor(d_mask_tensor, d_add_val, scale);

    dim3 Block(blockSize);
    dim3 Grid(gridSize);

    float milliseconds = 0.0f;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    for (int i = 0; i < 1000; ++i)
        FusedBaisAdd<bias_size><<<Grid, Block>>>(functor, dx, dy, d_bias, n, bias_size);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(y, dy, sizeof(float) * n, hipMemcpyDeviceToHost);

    bool isRight = CheckRight(y, groudTruth, n);
    if (isRight)
        printf("结果正确\n");
    else
        printf("结果错误\n");    

    printf("it costs %f s \n", milliseconds/1000);

    hipFree(dx);
    hipFree(dy);
    hipFree(d_bias);
    hipFree(d_add_val);
    hipFree(d_mask_tensor);
    free(x);
    free(y);
    free(bias);
    free(groudTruth);
    delete mask_tensor;
    mask_tensor = nullptr;
    delete add_val;
    add_val = nullptr;
}