
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N 100000000
#define BLOCK_SIZE 256
#define GRID_SIZE 32
#define topk 10

__managed__ int source[N];
__managed__ int gpu_result[topk];
__managed__ int _1_pass_result[topk * GRID_SIZE];

// Insert a data into an array (containing k values), still keeping the order of the array from max to min
__device__ __host__ void insert_value(int *array, int k, int data)
{
    for(int i=0; i<k; i++)
    {
        if(array[i] == data)
        {
            return;
        }
    }
    if(data < array[k-1])
    {
        return;
    }
    //19, 18, 17, 16,.........4, 3, 2, 1, 0
    for(int i = k-2; i>=0; i--)
    {
        if(data > array[i])
        {
            array[i + 1] = array[i];
        }
        else
        {
            array[i+1] = data;
            return;
        }
    }
    
    array[0] = data;
}

__global__ void gpu_topk(int *input, int *output, int length, int k)
{
    // my code 
    int array[topk]; 
    for (int i = 0; i < topk; i++)
    {
        array[i] = INT_MIN;
    }
    
    __shared__ int sh [BLOCK_SIZE * topk];
    
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < length; idx += gridDim.x * blockDim.x)
    {
        insert_value(array, k, input[idx]);
    }
    for (int j = 0; j < topk; j++)
    {
        sh[topk * threadIdx.x + j] = array[j];
    }
    __syncthreads();

    for(int i = BLOCK_SIZE / 2; i >= 1; i /= 2)
    {
        if (threadIdx.x < i)
        {
            for (int m = 0; m < topk; m++)
            {
                insert_value(array, topk, sh[topk * (threadIdx.x + i) + m]);
            }
        }            
        __syncthreads();
        if(threadIdx.x < i)
        {
            for (int m = 0; m < topk; m++)
            {
                sh[topk* threadIdx.x + m] = array[m];
            }
        }
        __syncthreads();
    }
    
    if (blockIdx.x * blockDim.x < length)
    {
        if (threadIdx.x == 0)
        {
            for (int m = 0; m < topk; m++)
            {
                output[topk * blockIdx.x + m] = sh[m];
            }
        }
    }

}

void cpu_topk(int *input, int *output, int length, int k)
{
    for(int i =0; i< length; i++)
    {
        insert_value(output, k, input[i]);
    }
}

int main()
{
    printf("Init source data...........\n");
    for(int i=0; i<N; i++)
    {
        source[i] = rand();
    }

    printf("Complete init source data.....\n");
    hipEvent_t start, stop_gpu, stop_cpu;
    hipEventCreate(&start);
    hipEventCreate(&stop_gpu);
    hipEventCreate(&stop_cpu);

    hipEventRecord(start);
    hipEventSynchronize(start);
    printf("GPU Run **************\n");
    for(int i =0; i<20; i++)
    {
        gpu_topk<<<GRID_SIZE, BLOCK_SIZE>>>(source, _1_pass_result, N, topk);

        gpu_topk<<<1, BLOCK_SIZE>>>(_1_pass_result, gpu_result, topk * GRID_SIZE, topk);
        // gpu_topk<<<1, BLOCK_SIZE>>>(source, gpu_result, N, topk);

        hipDeviceSynchronize();
    }
    printf("GPU Complete!!!\n");
    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);
    
    int cpu_result[topk] ={0};
    printf("CPU RUN***************\n");
    cpu_topk(source, cpu_result, N, topk);
    hipEventRecord(stop_cpu);
    hipEventSynchronize(stop_cpu);
    printf("CPU Complete!!!!!");

    float time_cpu, time_gpu;
    hipEventElapsedTime(&time_gpu, start, stop_gpu);
    hipEventElapsedTime(&time_cpu, stop_gpu, stop_cpu);

    bool error = false;
    for(int i =0; i<topk; i++)
    {
        printf("CPU top%d: %d; GPU top%d: %d;\n", i+1, cpu_result[i], i+1, gpu_result[i]);
        if(fabs(gpu_result[i] - cpu_result[i]) > 0)
        {
            error = true;
        }
    }
    printf("Result: %s\n", (error?"Error":"Pass"));
    printf("CPU time: %.2f; GPU time: %.2f\n", time_cpu, (time_gpu/20.0));
}




