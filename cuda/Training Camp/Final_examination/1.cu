#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "error.cuh"

#define N 3000 // Love u 3000 times!
#define BLOCK_SIZE 32

__managed__ int input_Matrix[N][N];
__managed__ int output_GPU[N][N];
__managed__ int output_CPU[N][N];
__global__ void gpu_ken(int input_M[N][N], int output_M[N][N])
{
    int x = blockIdx.x* blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x<N && y<N){
        if(input_M[y][x] > 100){
            output_M[y][x] = 0;
        }else{
            output_M[y][x] = input_M[y][x];
        }
    }

}
void cpu_ken(int input_M[N][N], int output_CPU[N][N])
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            if(input_M[i][j]>100)
            {
                output_CPU[i][j] = 0;
            }
            else
            {
                output_CPU[i][j] = input_Matrix[i][j];
            }
        }
    }
}

int main(int argc, char const* argv[])
{

    hipEvent_t start, stop_gpu;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop_gpu));


    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j)
        {

            input_Matrix[i][j] = rand() % 3001;
        }
    }
    cpu_ken(input_Matrix, output_CPU);

    CHECK(hipEventRecord(start));
    unsigned int grid_rows = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    printf("\n***********GPU RUN**************\n");
    gpu_ken<<<dimGrid, dimBlock >>>(input_Matrix, output_GPU);
    CHECK(hipDeviceSynchronize());
    CHECK(hipEventRecord(stop_gpu));
    CHECK(hipEventSynchronize(stop_gpu));

    float elapsed_time_gpu;
    CHECK(hipEventElapsedTime(&elapsed_time_gpu, start, stop_gpu));
    printf("Time_GPU = %g ms.\n", elapsed_time_gpu);

    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop_gpu));

    int ok = 1;
    printf("\n***********Check result**************\n");
    for (int i = 0; i < N; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            if (fabs(output_GPU[i][j] - output_CPU[i][j]) > (1.0e-10))
            {
                ok = 0;
                printf("cpu: %d; gpu: %d;\n", output_CPU[i][j], output_GPU[i][j]);
            }

        }
    }


    if (ok)
    {
        printf("Pass!!!\n");
    }
    else
    {
        printf("Error!!!\n");
    }

    // free memory
    return 0;
}