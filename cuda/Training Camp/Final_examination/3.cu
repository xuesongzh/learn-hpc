#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "error.cuh"

#define TILE_DIM 32   //Don't ask me why I don't set these two values to one
#define BLOCK_SIZE 32
#define N 3000 // for huanhuan, you know that!

__managed__ int input_M[N * N];      //input matrix & GPU result
int cpu_result[N * N];   //CPU result


//in-place matrix transpose
__global__ void ip_transpose(int* data)
{
    // my code 
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    
    
    __shared__ int tile_1[TILE_DIM][TILE_DIM+1];
    __shared__ int tile_2[TILE_DIM][TILE_DIM+1];
    
    if (blockIdx.y > blockIdx.x)
    {
        int dx = blockIdx.y * TILE_DIM + threadIdx.x;
        int dy = blockIdx.x * TILE_DIM + threadIdx.y;
        if(x<N && y<N)
        {
            tile_1[threadIdx.y][threadIdx.x] = data[y*N + x];
        }
        if( dx<N && dy<N)
        {
            tile_2[threadIdx.y][threadIdx.x] = data[dy*N + dx];
        }
        __syncthreads();
        if(dx<N && dy<N)
        {
            data[dy*N + dx] = tile_1[threadIdx.x][threadIdx.y];
        }
        if(x<N && y<N)
        {
            data[y*N + x] = tile_2[threadIdx.x][threadIdx.y];
        }
    }
    else if (blockIdx.y == blockIdx.x)
    {
        if(x<N && y<N)
        {
            tile_1[threadIdx.y][threadIdx.x] = data[y*N + x];
        }
        __syncthreads();
        if(x<N && y<N)
        {
            data[y*N + x] = tile_1[threadIdx.x][threadIdx.y];
        }
    }
}
void cpu_transpose(int* A, int* B)
{
    for (int j = 0; j < N; j++)
    {
        for (int i = 0; i < N; i++)
        {
            B[i * N + j] = A[j * N + i];
        }
    }
}

int main(int argc, char const* argv[])
{

    hipEvent_t start, stop_gpu;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop_gpu));


    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            input_M[i * N + j] = rand() % 1000;
        }
    }
    cpu_transpose(input_M, cpu_result);

    CHECK(hipEventRecord(start));
    unsigned int grid_rows = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    ip_transpose <<<dimGrid, dimBlock >>>(input_M);
    CHECK(hipDeviceSynchronize());
    CHECK(hipEventRecord(stop_gpu));
    CHECK(hipEventSynchronize(stop_gpu));

    float elapsed_time_gpu;
    CHECK(hipEventElapsedTime(&elapsed_time_gpu, start, stop_gpu));
    printf("Time_GPU = %g ms.\n", elapsed_time_gpu);

    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop_gpu));

    int ok = 1;
    for (int i = 0; i < N; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            if (fabs(input_M[i * N + j] - cpu_result[i * N + j]) > (1.0e-10))
            {
                ok = 0;
            }
        }
    }


    if (ok)
    {
        printf("Pass!!!\n");
    }
    else
    {
        printf("Error!!!\n");
    }

    return 0;
}